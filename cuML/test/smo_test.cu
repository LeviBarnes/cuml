#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "svm/workingset.h"
#include "svm/smosolver.h"
#include "svm/nonlinear.h"
#include <gtest/gtest.h>
#include <cuda_utils.h>
#include <test_utils.h>
#include <iostream>
#include <hipcub/hipcub.hpp>   

namespace ML {
namespace SVM {
using namespace MLCommon;

TEST(SmoSolverTestF, SelectWorkingSetTest) {
  WorkingSet<float> *ws;
  
  ws = new WorkingSet<float>(10);
  EXPECT_EQ(ws->GetSize(), 10);
  delete ws;
  
  ws = new WorkingSet<float>(100000);
  EXPECT_EQ(ws->GetSize(), 1024);
  delete ws;

  ws = new WorkingSet<float>(10, 4);
  EXPECT_EQ(ws->GetSize(), 4);
  
  float f_host[10] = {1, 3, 10, 4, 2, 8, 6, 5, 9, 7};
  float *f_dev;

  float y_host[10] = {-1, -1, -1, -1, -1, 1, 1, 1, 1, 1};
  float *y_dev;
  
  float C=1.5;
  
  float alpha_host[10] = {0, 0, 0.1, 0.2, 1.5, 0, 0.2, 0.4, 1.5, 1.5 };
  float *alpha_dev;  //   l  l  l/u  l/u    u  u  l/u  l/u  l    l
    
  int expected_idx[4] = {4, 3, 8, 2};
  allocate(f_dev, 10);
  allocate(y_dev, 10);
  allocate(alpha_dev, 10);
  updateDevice(f_dev, f_host, 10);
  updateDevice(y_dev, y_host, 10); 
  updateDevice(alpha_dev, alpha_host, 10);
  
  ws->Select(f_dev, alpha_dev, y_dev, C);
  int idx[4];
  updateHost(idx, ws->idx, 4);  
  for (int i=0; i<4; i++) {
    EXPECT_EQ(idx[i], expected_idx[i]);
  }
  CUDA_CHECK(hipFree(f_dev));
  CUDA_CHECK(hipFree(y_dev));
  CUDA_CHECK(hipFree(alpha_dev));
  delete ws;
}

TEST(SmoSolverTest, KernelCacheTest) {
    int n_rows = 4;
    int n_cols = 2;
    int n_ws = n_rows;
    
    float *x_dev;
    allocate(x_dev, n_rows*n_cols);
    int *ws_idx_dev;
    allocate(ws_idx_dev, n_ws);
    
    float x_host[] = { 1, 2, 3, 4, 5, 6, 7, 8};
    updateDevice(x_dev, x_host, n_rows*n_cols);
    
    int ws_idx_host[] = {0, 1, 2, 3};
    updateDevice(ws_idx_dev, ws_idx_host, n_ws);
    
    float tile_host[16];
    float tile_host_expected[] = {
      26, 32, 38, 44,
      32, 40, 48, 56,
      38, 48, 58, 68,
      44, 56, 68, 80
    };
    
    hipblasHandle_t cublas_handle;
    CUBLAS_CHECK(hipblasCreate(&cublas_handle));
    
    KernelCache<float> *cache = new KernelCache<float>(x_dev, n_rows, n_cols, n_ws, cublas_handle);
    float *tile_dev = cache->GetTile(ws_idx_dev);
    updateHost(tile_host, tile_dev, n_ws*n_rows);
    
    for (int i=0; i<n_ws*n_ws; i++) {
      EXPECT_EQ(tile_host[i], tile_host_expected[i])<< "First tile " << i;
    }
    
    // now check with selecting a subset of the rows
    delete cache;
    n_ws = 2;
    cache = new KernelCache<float>(x_dev, n_rows, n_cols, n_ws, cublas_handle);
    ws_idx_host[1] = 3; // i.e. ws_idx_host[] = {0,3}
    updateDevice(ws_idx_dev, ws_idx_host, n_ws);
    tile_dev = cache->GetTile(ws_idx_dev);
    updateHost(tile_host, tile_dev, n_ws*n_rows);
    
    float tile_expected2[] = {
      26, 32, 38, 44,
      44, 56, 68, 80
    };
    for (int i=0; i<n_ws*n_rows; i++) {
      EXPECT_EQ(tile_host[i], tile_expected2[i]) << "third tile " << i;
    }
    delete cache; 
    CUBLAS_CHECK(hipblasDestroy(cublas_handle));
    CUDA_CHECK(hipFree(x_dev));
    CUDA_CHECK(hipFree(ws_idx_dev));
}

template <typename math_t>
class LambdaContainer {
   //extended lambdas cannot be global scope nor can they be in the scope of a gtest
   //This seems to work.

   public:

   math_t (*polynomial_kernel)(math_t) = [] __device__ __host__ (math_t a) {return (1+a)*(1+a); };

};
TEST(SmoSolverTest, KernelCacheNonLinear) {
    int n_rows = 4;
    int n_cols = 2;
    int n_ws = n_rows;
    
    float *x_dev;
    allocate(x_dev, n_rows*n_cols);
    int *ws_idx_dev;
    allocate(ws_idx_dev, n_ws);
    
    float x_host[] = { 1, 2, 3, 4, 5, 6, 7, 8};
    updateDevice(x_dev, x_host, n_rows*n_cols);
    
    int ws_idx_host[] = {0, 1, 2, 3};
    updateDevice(ws_idx_dev, ws_idx_host, n_ws);
    
    float tile_host[16];
    float tile_host_expected[] = {
      26, 32, 38, 44,
      32, 40, 48, 56,
      38, 48, 58, 68,
      44, 56, 68, 80
    };
    
    hipblasHandle_t cublas_handle;
    CUBLAS_CHECK(hipblasCreate(&cublas_handle));
    
    //Polynomial kernel with exponent=2
    //auto nonlin = new polynomialKernel<float,int>(2);
    LambdaContainer<float> lambdas;
    auto nonlin = new generalKernel<float>(lambdas.polynomial_kernel);
    for (int z=0;z<16;z++) tile_host_expected[z] = (1+tile_host_expected[z])*(1+tile_host_expected[z]);
    KernelCache<float> *cache = 
           new KernelCache<float>(x_dev, n_rows, n_cols, n_ws, cublas_handle, nonlin);
    float *tile_dev = cache->GetTile(ws_idx_dev);
    updateHost(tile_host, tile_dev, n_ws*n_rows);
    
    for (int i=0; i<n_ws*n_ws; i++) {
      EXPECT_EQ(tile_host[i], tile_host_expected[i])<< "First tile " << i;
    }
    
    // now check with selecting a subset of the rows
    delete cache;
    n_ws = 2;
    cache = new KernelCache<float>(x_dev, n_rows, n_cols, n_ws, cublas_handle);
    ws_idx_host[1] = 3; // i.e. ws_idx_host[] = {0,3}
    updateDevice(ws_idx_dev, ws_idx_host, n_ws);
    tile_dev = cache->GetTile(ws_idx_dev);
    updateHost(tile_host, tile_dev, n_ws*n_rows);
    
    float tile_expected2[] = {
      26, 32, 38, 44,
      44, 56, 68, 80
    };
    for (int i=0; i<n_ws*n_rows; i++) {
      EXPECT_EQ(tile_host[i], tile_expected2[i]) << "third tile " << i;
    }
    delete cache; 
    CUBLAS_CHECK(hipblasDestroy(cublas_handle));
    CUDA_CHECK(hipFree(x_dev));
    CUDA_CHECK(hipFree(ws_idx_dev));
}

__global__ void init_training_vectors(float * x, int n_rows, int n_cols, int *ws_idx, int n_ws) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < n_rows * n_cols) {
      int i = tid % n_rows;
      int k = tid / n_rows;
      x [tid] = tid;
      if (k==0) {
          ws_idx[i] = i;
      }
    }
    
}
TEST(SmoSolverTest, KernelCacheLargeTest) {
    int n_rows = 10;
    int n_cols = 700;
    int n_ws = n_rows;
    
    float *x_dev;
    allocate(x_dev, n_rows*n_cols);
    int *ws_idx_dev;
    allocate(ws_idx_dev, n_ws);
    
    int TPB=256;
    init_training_vectors<<<ceildiv(n_rows*n_cols, TPB), TPB>>>(x_dev, n_rows, n_cols, ws_idx_dev, n_ws);
    CUDA_CHECK(hipPeekAtLastError());
    
    hipblasHandle_t cublas_handle;
    CUBLAS_CHECK(hipblasCreate(&cublas_handle));
    
    KernelCache<float> *cache = new KernelCache<float>(x_dev, n_rows, n_cols, n_ws, cublas_handle);
    float *tile_dev = cache->GetTile(ws_idx_dev);
    float *tile_host = new float[n_rows*n_cols];
    updateHost(tile_host, tile_dev, n_ws*n_rows);
    
    /*for (int i=0; i<n_ws*n_ws; i++) {
      EXPECT_EQ(tile_host[i], tile_host_expected[i])<< "First tile " << i;
    }*/
    
    delete cache; 
    delete[] tile_host;
    CUBLAS_CHECK(hipblasDestroy(cublas_handle));
    CUDA_CHECK(hipFree(x_dev));
    CUDA_CHECK(hipFree(ws_idx_dev));
}

// test a single iteration of the block solver
TEST(SmoSolverTest, SmoBlockSolveSingleTest) {
  int n_rows = 4;
  int n_cols = 2;
  int n_ws = n_rows;
    

  int *ws_idx_dev;
  allocate(ws_idx_dev, n_ws);
  float *y_dev;
  allocate(y_dev, n_rows);
  float *f_dev;
  allocate(f_dev, n_rows);
  float *alpha_dev;
  allocate(alpha_dev, n_rows, true);
  float *delta_alpha_dev;
  allocate(delta_alpha_dev, n_ws, true);
  float *kernel_dev;
  allocate(kernel_dev, n_ws*n_rows);
  float *return_buff_dev;
  allocate(return_buff_dev, 2);
    
  int ws_idx_host[] = {0, 1, 2, 3};
  updateDevice(ws_idx_dev, ws_idx_host, n_ws);
  
  float y_host[] = {1, 1, -1, -1};
  updateDevice(y_dev, y_host, n_rows);

  float f_host[] = {0.4, 0.3, 0.5, 0.1};
  updateDevice(f_dev, f_host, n_rows);

  float kernel_host[] = {
      26, 32, 38, 44,
      32, 40, 48, 56,
      38, 48, 58, 68,
      44, 56, 68, 80
  };
  
  updateDevice(kernel_dev, kernel_host, n_ws*n_rows);

  SmoBlockSolve<float, 1024><<<1, n_ws>>>(y_dev, n_rows, alpha_dev, n_ws, 
      delta_alpha_dev, f_dev, kernel_dev, ws_idx_dev,
      1.5f, 1e-3f, return_buff_dev, 1);
  
  CUDA_CHECK(hipPeekAtLastError());
  
  float return_buff[2];
  updateHost(return_buff, return_buff_dev, 2);
  EXPECT_FLOAT_EQ(return_buff[0], 0.2f) << return_buff[0];
  EXPECT_EQ(return_buff[1], 1) << "Number of iterations ";
  
  float host_alpha[4], host_dalpha[4];
  updateHost(host_alpha, alpha_dev, n_rows);
  updateHost(host_dalpha, delta_alpha_dev, n_ws);
  
  for (int i=0; i<n_ws; i++) {
      EXPECT_FLOAT_EQ(y_host[i]*host_alpha[i], host_dalpha[i]) << "alpha and delta alpha " << i;
  }
  float alpha_expected[] = {0, 0.1f, 0.1f, 0};
  for (int i=0; i<n_rows; i++) {
      EXPECT_FLOAT_EQ(host_alpha[i], alpha_expected[i]) << "alpha " << i;
  }
  
  // now check if updateF works
  SmoSolver<float> smo;
  hipblasHandle_t cublas_handle;
  CUBLAS_CHECK(hipblasCreate(&cublas_handle));

  smo.UpdateF(f_dev, n_rows, delta_alpha_dev, n_ws, kernel_dev, cublas_handle);
  updateHost(f_host, f_dev, n_rows);
  float f_host_expected[] = {-0.2, -0.5, -0.5, -1.1};
  for (int i=0; i<n_rows; i++) {
      EXPECT_FLOAT_EQ(f_host[i], f_host_expected[i]) << "UpdateF " << i;
  }   
  CUBLAS_CHECK(hipblasDestroy(cublas_handle));
  CUDA_CHECK(hipFree(y_dev));
  CUDA_CHECK(hipFree(f_dev));
  CUDA_CHECK(hipFree(ws_idx_dev));
  CUDA_CHECK(hipFree(alpha_dev));
  CUDA_CHECK(hipFree(delta_alpha_dev));
  CUDA_CHECK(hipFree(kernel_dev));
  CUDA_CHECK(hipFree(return_buff_dev));
}


TEST(SmoSolverTest, SmoBlockSolveTest) {
  int n_rows = 6;
  int n_cols = 2;
  int n_ws = n_rows;
    
  float *x_dev;
  allocate(x_dev, n_rows*n_cols);
  int *ws_idx_dev;
  allocate(ws_idx_dev, n_ws);
  float *y_dev;
  allocate(y_dev, n_rows);
  float *f_dev;
  allocate(f_dev, n_rows);
  float *alpha_dev;
  allocate(alpha_dev, n_rows, true);
  float *delta_alpha_dev;
  allocate(delta_alpha_dev, n_ws, true);
  float *kernel_dev;
  allocate(kernel_dev, n_ws*n_rows);
  float *return_buff_dev;
  allocate(return_buff_dev, 2);
  
  float x_host[] = {1, 2, 1, 2, 1, 2,   1, 1, 2, 2, 3, 3};
  updateDevice(x_dev, x_host, n_rows*n_cols);
    
  int ws_idx_host[] = {0, 1, 2, 3, 4, 5};
  updateDevice(ws_idx_dev, ws_idx_host, n_ws);
  
  float y_host[] = {-1, -1, 1, -1, 1, 1};
  updateDevice(y_dev, y_host, n_rows);

  float f_host[] = {1, 1, -1, 1, -1, -1};
  updateDevice(f_dev, f_host, n_rows);

  
  float kernel_host[] = {
    2, 3, 3,  4,  4,  5,
    3, 5, 4,  6,  5,  7,
    3, 4, 5,  6,  7,  8,
    4, 6, 6,  8,  8, 10,
    4, 5, 7,  8, 10, 11,
    5, 7, 8, 10, 11, 13
  };
  
  updateDevice(kernel_dev, kernel_host, n_ws*n_rows);

  SmoBlockSolve<float, 1024><<<1, n_ws>>>(y_dev, n_rows, alpha_dev, n_ws, 
      delta_alpha_dev, f_dev, kernel_dev, ws_idx_dev,
      1.0f, 1e-3f, return_buff_dev);
  
  CUDA_CHECK(hipPeekAtLastError());
  float return_buff[2];
  updateHost(return_buff, return_buff_dev, 2);
  EXPECT_FLOAT_EQ(return_buff[0], 2.0f) << return_buff[0];
  EXPECT_LT(return_buff[1], 100) << return_buff[1];
  
  float host_alpha[6], host_dalpha[6];
  updateHost(host_alpha, alpha_dev, n_rows);
  updateHost(host_dalpha, delta_alpha_dev, n_ws);
  
  for (int i=0; i<n_ws; i++) {
      EXPECT_FLOAT_EQ(y_host[i]*host_alpha[i], host_dalpha[i]) << "alpha and delta alpha " << i;
  }
  float w[] = {0,0};
  
  float alpha_expected[] = {0.6f, 0, 1, 1, 0, 0.6f};
  //for C=10: {0.25f, 0, 2.25f, 3.75f, 0, 1.75f};
  float ay=0;
  for (int i=0; i<n_rows; i++) {
   //   EXPECT_FLOAT_EQ(host_alpha[i], alpha_expected[i]) << "alpha " << i;
      w[0] += x_host[i] * host_alpha[i] * y_host[i]; 
      w[1] += x_host[i + n_rows] * host_alpha[i] * y_host[i];
      ay += host_alpha[i] * y_host[i];
  }
  EXPECT_FLOAT_EQ(ay, 0.0);
  EXPECT_FLOAT_EQ(w[0], -0.4);
  EXPECT_FLOAT_EQ(w[1],  1.2);
  // for C=10
  //EXPECT_FLOAT_EQ(w[0], -2.0);
  //EXPECT_FLOAT_EQ(w[1],  2.0);
  CUDA_CHECK(hipFree(x_dev));
  CUDA_CHECK(hipFree(y_dev));
  CUDA_CHECK(hipFree(f_dev));
  CUDA_CHECK(hipFree(ws_idx_dev));
  CUDA_CHECK(hipFree(alpha_dev));
  CUDA_CHECK(hipFree(delta_alpha_dev));
  CUDA_CHECK(hipFree(kernel_dev));
  CUDA_CHECK(hipFree(return_buff_dev));
}


TEST(SmoSolverTest, GetResultsTest) {
  int n_rows = 6;
  int n_cols = 2;
    
  float *x_dev;
  allocate(x_dev, n_rows*n_cols);

    
  float x_host[] = { 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12};
  updateDevice(x_dev, x_host, n_rows*n_cols);

  float *y_dev;
  allocate(y_dev, n_rows);
  
  float *alpha_dev;
  allocate(alpha_dev, n_rows);
  float y_host[] = {1, 1, 1, -1, -1, -1};
  updateDevice(y_dev, y_host, n_rows);

  float alpha_host[] = {0.0, 0.5, 0.5, 0, 1.0, 0,0};
  updateDevice(alpha_dev, alpha_host, n_rows);

  SmoSolver<float> smo;
  
  float *dual_coefs;
  int n_coefs;
  int *idx;
  float *x_support;
  float b;
  hipblasHandle_t cublas_handle;
  CUBLAS_CHECK(hipblasCreate(&cublas_handle));


  smo.GetResults(x_dev, n_rows, n_cols, y_dev, alpha_dev, &dual_coefs, &n_coefs, &idx, 
                 &x_support, &b, cublas_handle);
  
  ASSERT_EQ(n_coefs, 3);

  
  float dual_coefs_host[3];
  updateHost(dual_coefs_host, dual_coefs, n_coefs);
  float dual_coefs_exp[] = { 0.5, 0.5, -1.0 };
  for (int i=0; i<n_coefs; i++) {
    EXPECT_FLOAT_EQ(dual_coefs_host[i], dual_coefs_exp[i]) << "dual coeff idx " << i;
  }

  int idx_host[3];
  updateHost(idx_host, idx, n_coefs);
  float idx_exp[] = { 1, 2, 4 };
  for (int i=0; i<n_coefs; i++) {
    EXPECT_EQ(idx_host[i], idx_exp[i]) << "idx " << i;
  }
 
  float x_support_host[6];
  updateHost(x_support_host, x_support, n_coefs * n_cols);
  float x_support_exp[] = { 2, 3, 5,  8, 9, 11 };
  for (int i=0; i<n_coefs*n_cols; i++) {
    EXPECT_FLOAT_EQ(x_support_host[i], x_support_exp[i]) << "dual coeff idx " << i;
  }

  if (n_coefs > 0) {
    CUDA_CHECK(hipFree(dual_coefs));
    CUDA_CHECK(hipFree(idx));
    CUDA_CHECK(hipFree(x_support));
  }
  
  CUBLAS_CHECK(hipblasDestroy(cublas_handle));  
  CUDA_CHECK(hipFree(x_dev));
  CUDA_CHECK(hipFree(y_dev));
  CUDA_CHECK(hipFree(alpha_dev));
}


TEST(SmoSolverTest, SmoUpdateFTest) {
  int n_rows = 6;
  int n_cols = 2;
  int n_ws = 2;
    
  float *kernel_dev;
  allocate(kernel_dev, n_rows*n_ws);
  
  float *f_dev;
  allocate(f_dev, n_rows, true);
  
  float *delta_alpha_dev;
  allocate(delta_alpha_dev, n_ws);
  
  float kernel_host[] = {
    3, 5, 4,  6,  5,  7,
    4, 5, 7,  8, 10, 11
  };
  updateDevice(kernel_dev, kernel_host, n_ws*n_rows);
  
  float delta_alpha_host[] = {-0.1f, 0.1f};
  updateDevice(delta_alpha_dev, delta_alpha_host, n_ws);

  SmoSolver<float> smo(1, 0.001);

  hipblasHandle_t cublas_handle;
  CUBLAS_CHECK(hipblasCreate(&cublas_handle));

  smo.UpdateF(f_dev, n_rows, delta_alpha_dev, n_ws, kernel_dev, cublas_handle);
  
  float f_host[6];
  updateHost(f_host, f_dev, n_rows);
  
  float f_host_expected[] = {0.1f, 7.4505806e-9f, 0.3f, 0.2f, 0.5f, 0.4f};
  for (int i=0; i<n_rows; i++) {
      EXPECT_FLOAT_EQ(f_host[i], f_host_expected[i]) << "UpdateF " << i;
  }   

  CUDA_CHECK(hipFree(delta_alpha_dev));
  CUDA_CHECK(hipFree(kernel_dev));
  CUDA_CHECK(hipFree(f_dev));
}

TEST(SmoSolverTest, SmoSolveTest) {
  int n_rows = 6;
  int n_cols = 2;
  int n_ws = n_rows;
    
  float *x_dev;
  allocate(x_dev, n_rows*n_cols);
  float *y_dev;
  allocate(y_dev, n_rows);
  
  float x_host[] = {1, 2, 1, 2, 1, 2,   1, 1, 2, 2, 3, 3};
  updateDevice(x_dev, x_host, n_rows*n_cols);
  
  float y_host[] = {-1, -1, 1, -1, 1, 1};
  updateDevice(y_dev, y_host, n_rows);

  SmoSolver<float> smo(1, 0.001);
  
  float *dual_coefs;
  int n_coefs;
  int *idx;
  float *x_support;
  float b;
  hipblasHandle_t cublas_handle;
  CUBLAS_CHECK(hipblasCreate(&cublas_handle));
  
  smo.Solve(x_dev, n_rows, n_cols, y_dev, &dual_coefs, &n_coefs, &x_support, &idx, &b, 
            cublas_handle, 100,1);
  
  ASSERT_EQ(n_coefs, 4);
  
  float dual_coefs_host[4];
  updateHost(dual_coefs_host, dual_coefs, n_coefs);  
  
  float dual_coefs_exp[] = { -0.6, 1, -1, 0.6 };
  float ay = 0;
  for (int i=0; i<n_coefs; i++) {
    EXPECT_FLOAT_EQ(dual_coefs_host[i], dual_coefs_exp[i]) << "dual coeff idx " << i;
    ay += dual_coefs_host[i];
  }
  
  // \sum \alpha_i y_i = 0
  EXPECT_LT(abs(ay), 1.0e-6f);
  
  int idx_host[4];
  updateHost(idx_host, idx, n_coefs);
  float idx_exp[] = { 0, 2, 3, 5 };
  for (int i=0; i<n_coefs; i++) {
    EXPECT_EQ(idx_host[i], idx_exp[i]) << "idx " << i;
  }
 
  float x_support_host[8];
  updateHost(x_support_host, x_support, n_coefs * n_cols);
  float x_support_exp[] = { 1, 1, 2, 2,  1, 2, 2, 3};
  for (int i=0; i<n_coefs*n_cols; i++) {
    EXPECT_FLOAT_EQ(x_support_host[i], x_support_exp[i]) << "dual coeff idx " << i;
  }
  
  float w[] = {0,0};
  
  for (int i=0; i<n_coefs; i++) {
      w[0] += x_support_host[i] * dual_coefs_host[i]; 
      w[1] += x_support_host[i + n_coefs] * dual_coefs_host[i];      
  }
  EXPECT_FLOAT_EQ(w[0], -0.4);
  EXPECT_FLOAT_EQ(w[1],  1.2);
  
  EXPECT_FLOAT_EQ(b, -1.8);
  
  CUBLAS_CHECK(hipblasDestroy(cublas_handle));
  if (n_coefs > 0) {
    CUDA_CHECK(hipFree(dual_coefs));
    CUDA_CHECK(hipFree(idx));
    CUDA_CHECK(hipFree(x_support));
  }
  CUDA_CHECK(hipFree(x_dev));
  CUDA_CHECK(hipFree(y_dev));
}

TEST(SmoSolverTest, SmoSolveTestLargeC) {
  int n_rows = 6;
  int n_cols = 2;
  int n_ws = n_rows;
    
  float *x_dev;
  allocate(x_dev, n_rows*n_cols);
  float *y_dev;
  allocate(y_dev, n_rows);
  
  float x_host[] = {1, 2, 1, 2, 1, 2,   1, 1, 2, 2, 3, 3};
  updateDevice(x_dev, x_host, n_rows*n_cols);
  
  float y_host[] = {-1, -1, 1, -1, 1, 1};
  updateDevice(y_dev, y_host, n_rows);

  float epsilon = 0.001;
  SmoSolver<float> smo(100, epsilon);
  
  float *dual_coefs;
  int n_coefs;
  int *idx;
  float *x_support;
  float b;
  hipblasHandle_t cublas_handle;
  CUBLAS_CHECK(hipblasCreate(&cublas_handle));
  smo.Solve(x_dev, n_rows, n_cols, y_dev, &dual_coefs, &n_coefs, &x_support, &idx, &b, 
            cublas_handle, 100, 1);
  
  ASSERT_LE(n_coefs, 4);
  
  float dual_coefs_host[4];
  updateHost(dual_coefs_host, dual_coefs, n_coefs);  
  
  float dual_coefs_exp[] = { -2, 4, -2, 0, 0 };
  float ay = 0;
  for (int i=0; i<n_coefs; i++) {
   // EXPECT_FLOAT_EQ(dual_coefs_host[i], dual_coefs_exp[i]) << "dual coeff idx " << i;
    ay += dual_coefs_host[i];
  }
  // \sum \alpha_i y_i = 0
  EXPECT_LT(abs(ay), 1.0e-6f);
  
  int idx_host[4];
  updateHost(idx_host, idx, n_coefs);
  float idx_exp[] = { 0, 2, 3 };
  for (int i=0; i<n_coefs; i++) {
   // EXPECT_EQ(idx_host[i], idx_exp[i]) << "idx " << i;
  }
 
  float x_support_host[8];
  updateHost(x_support_host, x_support, n_coefs * n_cols);
  float x_support_exp[] = { 1, 1, 2,  1, 2, 2, 0,0};
  for (int i=0; i<n_coefs*n_cols; i++) {
  //  EXPECT_FLOAT_EQ(x_support_host[i], x_support_exp[i]) << "dual coeff idx " << i;
  }
  
  
  float w[] = {0,0};
  
  for (int i=0; i<n_coefs; i++) {
      w[0] += x_support_host[i] * dual_coefs_host[i]; 
      w[1] += x_support_host[i + n_coefs] * dual_coefs_host[i];      
  }
  // for linear problems it should be unique 
  EXPECT_LT(abs(w[0] - (-2)), epsilon);
  EXPECT_LT(abs(w[1] - 2), epsilon);
 
  EXPECT_FLOAT_EQ(b, -1.0f);
  
  CUBLAS_CHECK(hipblasDestroy(cublas_handle));
  if (n_coefs > 0) {
    CUDA_CHECK(hipFree(dual_coefs));
    CUDA_CHECK(hipFree(idx));
    CUDA_CHECK(hipFree(x_support));
  }
  CUDA_CHECK(hipFree(x_dev));
  CUDA_CHECK(hipFree(y_dev));
}

/*TEST_F(SmoSolverTestF, SelectWorkingSetTest) {
  ASSERT_LT(1, 2);
}*/

}; // end namespace SVM
}; // end namespace ML
