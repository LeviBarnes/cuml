#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include "linalg/gemm.h"
#include "linalg/cutlass_wrappers.h"


namespace MLCommon {
namespace LinAlg {

template <typename T>
__global__ void fillKernel(T *arr, T val, int N) {
  const int stride = blockDim.x * gridDim.x;
  int tid = threadIdx.x + (blockIdx.x * blockDim.x);
  for (int i = tid; i < N; i += stride)
    arr[i] = val;
}

template <typename T, int NTHREADS = 256, int NITEMS = 4>
void fill(T *arr, T val, int N) {
  const int nblks = ceildiv<int>(N, NTHREADS * NITEMS);
  fillKernel<T><<<nblks, NTHREADS>>>(arr, val, N);
  CUDA_CHECK(hipPeekAtLastError());
}

class LambdaContainer {

   public:
   int (*null_init) ( cutlass::gemm::LinearScaling<float>::Params &) = 
              [] __host__ __device__ (cutlass::gemm::LinearScaling<float>::Params &p) {return 0;};
   float (*times_two) ( float, int) = 
              [] __host__ __device__ (float a, int inx) { return 10.f;};

   LambdaContainer() { }
};

#ifndef __CUDACC_EXTENDED_LAMBDA__
#error "compile with --expt-extended-lamdba"
#endif
TEST(Gemm, Gemm_w_Epilogue) {
  float *A, *B, *C, *D;
  int M = 128, N = 128, K = 64;
  CUDA_CHECK(hipMalloc((void **)&A, sizeof(float) * M * K));
  fill(A, 1.f, M * K);
  CUDA_CHECK(hipMalloc((void **)&B, sizeof(float) * K * N));
  fill(B, 0.5f, K * N);
  CUDA_CHECK(hipMalloc((void **)&C, sizeof(float) * M * N));
  fill(C, 2.f, M * N);
  CUDA_CHECK(hipMalloc((void **)&D, sizeof(float) * M * N));
  CUDA_CHECK(hipMemset(D, 0, sizeof(float) * M * N));
  gemm<float, float, float, cutlass::Shape<8, 128, 128>>(
    HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, 0.5f, B, N, A, K, 0.5f, C, N, D,
    LambdaContainer().null_init, LambdaContainer().times_two);
  float *hD = new float[M * N];
  updateHost<float>(hD, D, M * N);
  for (int i = 0; i < M * N; ++i) {
    ASSERT_FLOAT_EQ(0.5f * K + 2.f, hD[i]) << " @hD[" << i << "]";
  }
  delete[] hD;
  CUDA_CHECK(hipFree(A));
  CUDA_CHECK(hipFree(B));
  CUDA_CHECK(hipFree(C));
  CUDA_CHECK(hipFree(D));
}

TEST(Gemm, Gemm_128x128x8) {
  float *A, *B, *C, *D;
  int M = 128, N = 128, K = 64;
  CUDA_CHECK(hipMalloc((void **)&A, sizeof(float) * M * K));
  fill(A, 1.f, M * K);
  CUDA_CHECK(hipMalloc((void **)&B, sizeof(float) * K * N));
  fill(B, 0.5f, K * N);
  CUDA_CHECK(hipMalloc((void **)&C, sizeof(float) * M * N));
  fill(C, 2.f, M * N);
  CUDA_CHECK(hipMalloc((void **)&D, sizeof(float) * M * N));
  CUDA_CHECK(hipMemset(D, 0, sizeof(float) * M * N));
  gemm<float, float, float, cutlass::Shape<8, 128, 128>>(
    HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, 1.f, B, N, A, K, 1.f, C, N, D);
  float *hD = new float[M * N];
  updateHost<float>(hD, D, M * N);
  for (int i = 0; i < M * N; ++i) {
    ASSERT_FLOAT_EQ(0.5f * K + 2.f, hD[i]) << " @hD[" << i << "]";
  }
  delete[] hD;
  CUDA_CHECK(hipFree(A));
  CUDA_CHECK(hipFree(B));
  CUDA_CHECK(hipFree(C));
  CUDA_CHECK(hipFree(D));
}

} // end namespace LinAlg
} // end namespace MLCommon
